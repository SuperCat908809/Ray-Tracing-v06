#include "hip/hip_runtime.h"
#include <string>
#include <assert.h>
#include <stdexcept>
#include <iostream>
#include <hip/hip_runtime.h>
#include <>

#include <glm/glm.hpp>
#include <stb/stb_image_write.h>

#define _MISS_DIST 1e9f

struct Ray {
	glm::vec3 o{ 0,0,0 }, d{ 0,0,1 };
	float t{ _MISS_DIST };

	__host__ __device__ glm::vec3 at(float t) const { return o + d * t; }
};

struct TraceRecord {
	glm::vec3 n{ 0,1,0 };
};

struct Sphere {
	glm::vec3 origin{ 0,0,0 };
	float radius{ 1 };

	__host__ __device__ bool Trace(Ray& ray, TraceRecord& rec) const {
		glm::vec3 oc = origin - ray.o;

		float a = glm::dot(ray.d, ray.d);
		float hb = glm::dot(ray.d, oc);
		float c = glm::dot(oc, oc) - radius * radius;
		float d = hb * hb - a * c;
		if (d <= 0) return false;

		d = sqrtf(d);
		float t = (-hb - d) / a;
		if (t < 1e-6f || t > ray.t) {
			t = (-hb + d) / a;
			if (t < 1e-6f || t > ray.t) return false;
		}

		ray.t = t;
		rec.n = glm::normalize(ray.at(t) - origin);
	}
};

struct LaunchParams {
	uint32_t render_width{};
	uint32_t render_height{};
	glm::vec4* output_buffer{};
};

__global__ void kernel(LaunchParams p) {
	int x = blockDim.x * blockIdx.x + threadIdx.x;
	int y = blockDim.y * blockIdx.y + threadIdx.y;
	if (x >= p.render_width || y >= p.render_height) return;

	int gid = y * p.render_width + x;
	float u = x / (p.render_width - 1.0f);
	float v = y / (p.render_height - 1.0f);

#if 0
	p.output_buffer[gid] = glm::vec4(u, v, 0, 1);
#elif 1
	glm::vec3 o(0, 0, -4);
	glm::vec3 hori(2, 0, 0);
	glm::vec3 vert(0, 2, 0);
	glm::vec3 llc = (hori + vert) * -0.5f + glm::vec3(0, 0, 1);

	Ray ray{};
	ray.o = o;
	ray.d = llc + hori * u + vert * v;

	float t = glm::normalize(ray.d).y * 0.5f + 0.5f;
	glm::vec4 output_color = (1 - t) * glm::vec4(0.1f, 0.2f, 0.4f, 1.0f) + t * glm::vec4(0.9f, 0.9f, 0.99f, 1.0f);

	p.output_buffer[gid] = output_color;
#else
	Sphere sphere{};
	sphere.origin = glm::vec3(0);
	sphere.radius = 1;

	TraceRecord rec{};
	glm::vec4 output_color{};
	if (sphere.Trace(ray, rec)) {
		output_color = glm::vec4(rec.n * 0.5f + 0.5f, 1.0f);
	}
	else {
		float t = glm::normalize(ray.d).y * 0.5f + 0.5f;
		output_color = (1 - t) * glm::vec4(0.1f, 0.2f, 0.4f, 1.0f) + t * glm::vec4(0.9f, 0.9f, 0.99f, 1.0f);
	}

	p.output_buffer[gid] = output_color;
#endif
}

#define CUDA_CHECK(func) cudaAssert(func, #func, __FILE__, __LINE__)
#define CUDA_ASSERT(func) try { CUDA_CHECK(func); } catch (const std::runtime_error& e) { assert(0); }
inline void cudaAssert(hipError_t code, const char* func, const char* file, const int line) {
	if (code != hipSuccess) {
		fprintf(stderr, "GPU assert: %s %s\n%s %d\n%s :: %s",
			hipGetErrorName(code), func,
			file, line,
			hipGetErrorName(code), hipGetErrorString(code)
		);
		throw std::runtime_error(hipGetErrorString(code));
	}
}

int main() {

	LaunchParams p{};
	p.render_width = 1280;
	p.render_height = 720;

	glm::vec4* h_framebuffer{};
	glm::vec4* d_framebuffer{};

	CUDA_ASSERT(hipHostMalloc(&h_framebuffer, sizeof(glm::vec4) * p.render_width * p.render_height));
	CUDA_ASSERT(hipMalloc(&d_framebuffer, sizeof(glm::vec4) * p.render_width * p.render_height));

	p.output_buffer = d_framebuffer;

	dim3 threads{ 8, 8, 1 };
	//dim3 blocks = dim3((p.render_width + threads.x - 1) / threads.x, (p.render_height + threads.y - 1) / threads.y, 1);
	dim3 blocks{ 0,0,1 };
	blocks.x = (p.render_width + threads.x - 1) / threads.x;
	blocks.y = (p.render_height + threads.y - 1) / threads.y;
	kernel<<<blocks, threads>>>(p);
	CUDA_ASSERT(hipPeekAtLastError());
	CUDA_ASSERT(hipDeviceSynchronize());


	CUDA_ASSERT(hipMemcpy(h_framebuffer, d_framebuffer, sizeof(glm::vec4) * p.render_width * p.render_height, hipMemcpyDeviceToHost));

	const char* output_path = "../renders/test_002.png";
	uint8_t* output_image_data = new uint8_t[p.render_width * p.render_height * 4];
	for (int i = 0; i < p.render_width * p.render_height; i++) {
		output_image_data[i * 4 + 0] = static_cast<uint8_t>(h_framebuffer[i][0] * 255.999f);
		output_image_data[i * 4 + 1] = static_cast<uint8_t>(h_framebuffer[i][1] * 255.999f);
		output_image_data[i * 4 + 2] = static_cast<uint8_t>(h_framebuffer[i][2] * 255.999f);
		output_image_data[i * 4 + 3] = static_cast<uint8_t>(h_framebuffer[i][3] * 255.999f);
	}

	stbi_flip_vertically_on_write(true);
	stbi_write_png(output_path, p.render_width, p.render_height, 4, output_image_data, sizeof(uint8_t) * p.render_width * 4);
	delete[] output_image_data;


	CUDA_ASSERT(hipHostFree(h_framebuffer));
	CUDA_ASSERT(hipFree(d_framebuffer));

	CUDA_ASSERT(hipDeviceReset());

	return 0;
}