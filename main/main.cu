#define _USE_MATH_DEFINES
#include <math.h>
#include <string>
using namespace std::string_literals;
#include <assert.h>
#include <stdexcept>
#include <iostream>
#include <hip/hip_runtime.h>
#include <>

#include <glm/glm.hpp>
#include <stb/stb_image_write.h>

#include "cu_rtCommon.cuh"
#include "cu_Geometry.cuh"
#include "cu_Cameras.cuh"

#include "Renderer.cuh"
#include "FirstApp.cuh"

int main() {

	{

		dAbstract<Material> mat_ptr2 = dAbstract<Material>::MakeAbstract<MetalAbstract>(glm::vec3(1.0f), 0.05f);

		FirstApp app = FirstApp::MakeApp();
		app.Run();
	}

	CUDA_ASSERT(hipDeviceReset());

	return 0;
}