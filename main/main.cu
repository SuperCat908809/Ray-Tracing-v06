#include "hip/hip_runtime.h"
#define _USE_MATH_DEFINES
#include <math.h>
#include <string>
using namespace std::string_literals;
#include <assert.h>
#include <stdexcept>
#include <iostream>
#include <hip/hip_runtime.h>
#include <>

#include <glm/glm.hpp>
#include <stb/stb_image_write.h>

#include "cu_rtCommon.cuh"
#include "cu_geometry.cuh"
#include "cu_Cameras.cuh"

struct LaunchParams {
	uint32_t render_width{};
	uint32_t render_height{};
	PinholeCamera cam{};
	Sphere sphere{};
	glm::vec4* output_buffer{};
};

__global__ void kernel(LaunchParams p) {
	int x = blockDim.x * blockIdx.x + threadIdx.x;
	int y = blockDim.y * blockIdx.y + threadIdx.y;
	if (x >= p.render_width || y >= p.render_height) return;

	int gid = y * p.render_width + x;
	float u = x / (p.render_width - 1.0f) * 2 - 1;
	float v = y / (p.render_height - 1.0f) * 2 - 1;
	glm::vec2 ndc(u, v);

	Ray ray = p.cam.sample_ray(u, v);

	TraceRecord rec{};
	glm::vec4 output_color{};
	if (p.sphere.Trace(ray, rec)) {
		output_color = glm::vec4(rec.n * 0.5f + 0.5f, 1.0f);
	}
	else {
		float t = glm::normalize(ray.d).y * 0.5f + 0.5f;
		output_color = (1 - t) * glm::vec4(0.1f, 0.2f, 0.4f, 1.0f) + t * glm::vec4(0.9f, 0.9f, 0.99f, 1.0f);
	}

	p.output_buffer[gid] = output_color;
}

void write_renderbuffer_png(std::string filepath, uint32_t width, uint32_t height, glm::vec4* data) {
	uint8_t* output_image_data = new uint8_t[width * height * 4];
	for (int i = 0; i < width * height; i++) {
		output_image_data[i * 4 + 0] = static_cast<uint8_t>(data[i][0] * 255.999f);
		output_image_data[i * 4 + 1] = static_cast<uint8_t>(data[i][1] * 255.999f);
		output_image_data[i * 4 + 2] = static_cast<uint8_t>(data[i][2] * 255.999f);
		output_image_data[i * 4 + 3] = static_cast<uint8_t>(data[i][3] * 255.999f);
	}

	stbi_flip_vertically_on_write(true);
	stbi_write_png(filepath.c_str(), width, height, 4, output_image_data, sizeof(uint8_t) * width * 4);
	delete[] output_image_data;
}

int main() {

	LaunchParams p{};
	p.render_width = 1280;
	p.render_height = 720;

	glm::vec3 lookfrom(0, 0, 4);
	glm::vec3 lookat(0, 0, 0);
	glm::vec3 up(0, 1, 0);
	float fov = 90.0f;
	float aspect = p.render_width / (float)p.render_height;
	p.cam = PinholeCamera(lookfrom, lookat, up, fov, aspect);

	Sphere sphere{};
	sphere.origin = glm::vec3(0, 0, 0);
	sphere.radius = 1;
	p.sphere = sphere;

	glm::vec4* h_framebuffer{};
	glm::vec4* d_framebuffer{};

	CUDA_ASSERT(hipHostMalloc(&h_framebuffer, sizeof(glm::vec4) * p.render_width * p.render_height));
	CUDA_ASSERT(hipMalloc(&d_framebuffer, sizeof(glm::vec4) * p.render_width * p.render_height));

	p.output_buffer = d_framebuffer;

	dim3 threads{ 8, 8, 1 };
	dim3 blocks = dim3((p.render_width + threads.x - 1) / threads.x, (p.render_height + threads.y - 1) / threads.y, 1);
	kernel<<<blocks, threads>>>(p);
	CUDA_ASSERT(hipPeekAtLastError());
	CUDA_ASSERT(hipDeviceSynchronize());


	CUDA_ASSERT(hipMemcpy(h_framebuffer, d_framebuffer, sizeof(glm::vec4) * p.render_width * p.render_height, hipMemcpyDeviceToHost));

	write_renderbuffer_png("../renders/test_006.png"s, p.render_width, p.render_height, h_framebuffer);


	CUDA_ASSERT(hipHostFree(h_framebuffer));
	CUDA_ASSERT(hipFree(d_framebuffer));

	CUDA_ASSERT(hipDeviceReset());

	return 0;
}