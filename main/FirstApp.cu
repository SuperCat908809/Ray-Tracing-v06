#include "FirstApp.cuh"

#include <inttypes.h>
#include <string>
using namespace std::string_literals;
#include <iostream>
#include <vector>

#include <glm/glm.hpp>
#include <hip/hip_runtime.h>
#include <stb/stb_image_write.h>

#include "cuError.h"

#include "dobj.cuh"
#include "darray.cuh"
#include "dAbstracts.cuh"

#include "hittable.cuh"
#include "SphereHittable.cuh"
#include "HittableList.cuh"

#include "material.cuh"
#include "cu_Materials.cuh"

#include "cu_Cameras.cuh"
#include "Renderer.cuh"

#include "cuHostRND.h"


#if 0
class Scene1Factory {
public:

	class d_MatFactory {
	public:
		__device__ Material* operator()(size_t input) const {
			switch (input) {
			case 0: return new LambertianAbstract(glm::vec3(0.8f, 0.8f, 0.0f));
			case 1: return new LambertianAbstract(glm::vec3(0.1f, 0.2f, 0.5f));
			case 2: return new DielectricAbstract(glm::vec3(1.0f, 1.0f, 1.0f), 1.5f);
			case 3: return new      MetalAbstract(glm::vec3(0.8f, 0.6f, 0.2f), 0.0f);
			default: return nullptr;
			}
		}
	};

	class d_SphereFactory {
		Material** mat_ptrs{};
	public:
		__device__ d_SphereFactory(Material** ptr2) : mat_ptrs(ptr2) {}
		__device__ Hittable* operator()(size_t input) const {
			switch (input) {
			case 0: return new SphereHittable(glm::vec3( 0.0f, -100.5f, -1.0f), 100.0f, mat_ptrs[0]);
			case 1: return new SphereHittable(glm::vec3( 0.0f,    0.0f, -1.0f),   0.5f, mat_ptrs[1]);
			case 2: return new SphereHittable(glm::vec3(-1.0f,    0.0f, -1.0f),   0.5f, mat_ptrs[2]);
			case 3: return new SphereHittable(glm::vec3(-1.0f,    0.0f, -1.0f),  -0.4f, mat_ptrs[2]);
			case 4: return new SphereHittable(glm::vec3( 1.0f,    0.0f, -1.0f),   0.5f, mat_ptrs[3]);
			default: return nullptr;
			}
		}
	};

	Scene1Factory(
		dAbstractArray<Material>** materials,
		dAbstractArray<Hittable>** spheres,
		dAbstract<HittableList>** world_list
	) {
		dAbstract<d_MatFactory> matFact{};
		matFact.MakeOnDevice<d_MatFactory>();
		
		(*materials) = new dAbstractArray<Material>(4);
		(*materials)->MakeOnDeviceFactoryPtr<d_MatFactory>(4, 0, 0, matFact.getPtr());

		dAbstract<d_SphereFactory> sphereFact((*materials)->getDeviceArrayPtr());
		(*spheres) = new dAbstractArray<Hittable>(5);
		(*spheres)->MakeOnDeviceFactoryPtr<d_SphereFactory>(5, 0, 0, sphereFact.getPtr());

		(*world_list) = new dAbstract<HittableList>((*spheres)->getDeviceArrayPtr(), 5);
	}
};
#endif

#if 0
class SceneBook1FinaleFactory {

	//static std::vector<float> _makeNUniforms(size_t N, size_t seed) {
	//	hiprandGenerator_t gen;
	//	float* d_rnd_uniforms{};
	//	hipMalloc(&d_rnd_uniforms, sizeof(float) * N);

	//	hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_XORWOW);
	//	hiprandSetPseudoRandomGeneratorSeed(gen, seed);
	//	hiprandGenerateUniform(gen, d_rnd_uniforms, N);

	//	std::vector<float> rnds{};
	//	rnds.resize(N);
	//	hipMemcpy(rnds.data(), d_rnd_uniforms, sizeof(float) * N, hipMemcpyDeviceToHost);

	//	hiprandDestroyGenerator(gen);
	//	hipFree(d_rnd_uniforms);
	//	return rnds;
	//}

	template <typename T>
	static T* _copyToDevice(const std::vector<T>& v) {
		T* d_ptr{};
		CUDA_ASSERT(hipMalloc(&d_ptr, sizeof(T) * v.size()));
		CUDA_ASSERT(hipMemcpy(d_ptr, v.data(), sizeof(T) * v.size(), hipMemcpyHostToDevice));
		return d_ptr;
	}

	enum MatIdx { lambert, metal, dielec };
	struct LambertParams { glm::vec3 albedo; };
	struct MetalParams   { glm::vec3 albedo; float   fuzz; };
	struct DielecParams  { glm::vec3 albedo; float    ior; };
	struct SphereParams  { glm::vec3 origin; float radius; MatIdx mat_type; size_t mat_index; };

	void _makeSphere(int a, int b) {
	#define rnd (host_rnd.next())

		float choose_mat = rnd;
		glm::vec3 pos(a + 0.9f * rnd, 0.2f, b + 0.9f * rnd);

		if (glm::length(pos - glm::vec3(4, 0.2f, 0)) > 0.9f) {
			if (choose_mat < 0.8f) {
				// diffuse
				glm::vec3 albedo = glm::vec3(rnd, rnd, rnd) * glm::vec3(rnd, rnd, rnd);
				sphere_params.push_back({ pos, 0.2f, lambert, lambert_params.size() });
				lambert_params.push_back({ albedo });
			}
			else if (choose_mat < 0.95f) {
				// metal
				glm::vec3 albedo = glm::vec3(rnd, rnd, rnd) * 0.5f + 0.5f;
				float fuzz = rnd * 0.5f;
				sphere_params.push_back({ pos, 0.2f, metal, metal_params.size() });
				metal_params.push_back({ albedo, fuzz });
			}
			else {
				// glass
				sphere_params.push_back({ pos, 0.2f, dielec, dielec_params.size() });
				dielec_params.push_back({ glm::vec3(1.0f), 1.5f });
			}
		}
	}

	void _populateWorld() {
		// ground sphere
		sphere_params.push_back({ glm::vec3(0,-1000,0), 1000, lambert, lambert_params.size()});
		lambert_params.push_back({ glm::vec3(0.5f) });

		for (int a = -11; a < 11; a++) {
			for (int b = -11; b < 11; b++) {
				_makeSphere(a, b);
			}
		}

		sphere_params.push_back({ glm::vec3(0, 1, 0), 1.0f, dielec, dielec_params.size() });
		dielec_params.push_back({ glm::vec3(1.0f), 1.5f });

		sphere_params.push_back({ glm::vec3(-4, 1, 0), 1.0f, lambert, lambert_params.size() });
		lambert_params.push_back({ glm::vec3(0.4f, 0.2f, 0.1f) });

		sphere_params.push_back({ glm::vec3(4, 1, 0), 1.0f, metal, metal_params.size() });
		metal_params.push_back({ glm::vec3(0.7f, 0.6f, 0.5f), 0.0f });
	}

	std::vector<LambertParams> lambert_params{};
	std::vector<  MetalParams>   metal_params{};
	std::vector< DielecParams>  dielec_params{};
	std::vector< SphereParams>  sphere_params{};
	cuHostRND host_rnd{ 512, 1984 };

	SceneBook1FinaleFactory() = default;

public:

	class d_LambertFactory {
		LambertParams* p{};
	public:
		__host__ __device__ d_LambertFactory(LambertParams* p) : p(p) {}
		__device__ Material* operator()(size_t index) const {
			LambertParams& p2 = p[index];
			return new LambertianAbstract(p2.albedo);
		}
	};
	class d_MetalFactory {
		MetalParams* p{};
	public:
		__host__ __device__ d_MetalFactory(MetalParams* p) : p(p) {}
		__device__ Material* operator()(size_t index) const {
			MetalParams& p2 = p[index];
			return new MetalAbstract(p2.albedo, p2.fuzz);
		}
	};
	class d_DielecFactory {
		DielecParams* p{};
	public:
		__host__ __device__ d_DielecFactory(DielecParams* p) : p(p) {}
		__device__ Material* operator()(size_t index) const {
			DielecParams& p2 = p[index];
			return new DielectricAbstract(p2.albedo, p2.ior);
		}
	};
	class d_SphereFactory {
		SphereParams* p{};
		Material** mats;
		int lambert_offset{}, metal_offset{}, dielec_offset{};

		__device__ int _getMatOffset(MatIdx idx) const {
			switch (idx) {
			case lambert: return lambert_offset;
			case   metal: return   metal_offset;
			case  dielec: return  dielec_offset;
			}
		}

	public:
		__host__ __device__ d_SphereFactory(SphereParams* p, Material** mats, int lambert_offset, int metal_offset, int dielec_offset)
			: p(p), mats(mats), lambert_offset(lambert_offset), metal_offset(metal_offset), dielec_offset(dielec_offset) {}
		__device__ Hittable* operator()(size_t index) const {
			SphereParams& p2 = p[index];
			int mat_offset = p2.mat_index + _getMatOffset(p2.mat_type);
			return new SphereHittable(p2.origin, p2.radius, mats[mat_offset]);
		}
	};

	static _SceneDescription MakeScene() {
		SceneBook1FinaleFactory factory{};

		factory._populateWorld();

		LambertParams* d_lambert_params = _copyToDevice(factory.lambert_params);
		d_LambertFactory lambert_factory(d_lambert_params);
		d_LambertFactory* d_lambert_factory{ nullptr };
		CUDA_ASSERT(hipMalloc((void**)&d_lambert_factory, sizeof(d_LambertFactory)));
		CUDA_ASSERT(hipMemcpy(d_lambert_factory, &lambert_factory, sizeof(d_LambertFactory), hipMemcpyHostToDevice));

		MetalParams* d_metal_params = _copyToDevice(factory.metal_params);
		d_MetalFactory metal_factory(d_metal_params);
		d_MetalFactory* d_metal_factory{ nullptr };
		CUDA_ASSERT(hipMalloc((void**)&d_metal_factory, sizeof(d_MetalFactory)));
		CUDA_ASSERT(hipMemcpy(d_metal_factory, &metal_factory, sizeof(d_MetalFactory), hipMemcpyHostToDevice));

		DielecParams* d_dielec_params = _copyToDevice(factory.dielec_params);
		d_DielecFactory dielec_factory(d_dielec_params);
		d_DielecFactory* d_dielec_factory{ nullptr };
		CUDA_ASSERT(hipMalloc((void**)&d_dielec_factory, sizeof(d_DielecFactory)));
		CUDA_ASSERT(hipMemcpy(d_dielec_factory, &dielec_factory, sizeof(d_DielecFactory), hipMemcpyHostToDevice));

		size_t lambert_offset = 0;
		size_t   metal_offset = lambert_offset + factory.lambert_params.size();
		size_t  dielec_offset = metal_offset + factory.metal_params.size();

		dAbstractArray<Material> materials(factory.sphere_params.size());
		materials.MakeOnDeviceFactory<d_LambertFactory>(factory.lambert_params.size(), lambert_offset, 0, d_lambert_factory);
		materials.MakeOnDeviceFactory<d_MetalFactory  >(factory.  metal_params.size(),   metal_offset, 0,   d_metal_factory);
		materials.MakeOnDeviceFactory<d_DielecFactory >(factory. dielec_params.size(),  dielec_offset, 0,  d_dielec_factory);

		CUDA_ASSERT(hipFree(d_lambert_factory));
		CUDA_ASSERT(hipFree(d_metal_factory  ));
		CUDA_ASSERT(hipFree(d_dielec_factory ));

		CUDA_ASSERT(hipFree(d_lambert_params));
		CUDA_ASSERT(hipFree(d_metal_params  ));
		CUDA_ASSERT(hipFree(d_dielec_params ));


		SphereParams* d_sphere_params = _copyToDevice(factory.sphere_params);
		d_SphereFactory sphere_factory(d_sphere_params, materials.getDeviceArrayPtr(), lambert_offset, metal_offset, dielec_offset);
		d_SphereFactory* d_sphere_factory{ nullptr };
		CUDA_ASSERT(hipMalloc((void**)&d_sphere_factory, sizeof(d_SphereFactory)));
		CUDA_ASSERT(hipMemcpy(d_sphere_factory, &sphere_factory, sizeof(d_SphereFactory), hipMemcpyHostToDevice));

		dAbstractArray<Hittable> sphere_list(factory.sphere_params.size());
		sphere_list.MakeOnDeviceFactory<d_SphereFactory>(factory.sphere_params.size(), 0, 0, d_sphere_factory);

		CUDA_ASSERT(hipFree(d_sphere_factory));
		CUDA_ASSERT(hipFree(d_sphere_params));


		dAbstract<HittableList> world_list = dAbstract<HittableList>::MakeAbstract<HittableList>(sphere_list.getDeviceArrayPtr(), sphere_list.getLength());

		return _SceneDescription{
			std::move(materials),
			std::move(sphere_list),
			std::move(world_list)
		};
	}
};
#else

#if 0
class MaterialCollection {
	dAbstractArray<Material> material_ptrs;
	dmemory material_data;
};

class MaterialCollectionFactory {
public:
	enum MaterialID { LAMBERTIAN, METAL, DIELECTRIC };
	struct MaterialParams {
		union {
			struct { MaterialID matID; } matID;
			struct { MaterialID matID; glm::vec3 albedo;             } lambert;
			struct { MaterialID matID; glm::vec3 albedo; float fuzz; }   metal;
			struct { MaterialID matID; glm::vec3 albedo; float  ior; }  dielec;
		};

		__device__ Material* MakeMaterial(Material* location) const {
			switch (matID.matID) {
			case LAMBERTIAN: return new (location) LambertianAbstract(lambert.albedo             );
			case      METAL: return new (location)      MetalAbstract(  metal.albedo,  metal.fuzz);
			case DIELECTRIC: return new (location) DielectricAbstract( dielec.albedo, dielec.ior );
			}
		}
	};

	class MaterialFactory {
		MaterialParams* p;
		Material* reserved_location_root;
		size_t* reserved_locations;
	public:
		__host__ __device__ MaterialFactory(MaterialParams* p, Material* reserved_location_root, size_t* reserved_locations) :
			p(p),
			reserved_location_root(reserved_location_root),
			reserved_locations(reserved_locations) {}
		__device__ Material* operator()(size_t index) const { return p[index].MakeMaterial(reserved_location_root + reserved_locations[index]); }
	};

private:
	MaterialCollectionFactory(const MaterialCollectionFactory&) = delete;
	MaterialCollectionFactory& operator=(const MaterialCollectionFactory&) = delete;

	std::vector<MaterialParams> queued_materials;
public:


	/*
	==== ==== ==== ==== ==== ==== ==== ==== ==== ====
	== All constructed materials will be placed in the same buffer in device memory. This will be achived using the placement new operator.
	== On the host, when a material is queued, it should be given a pointer offset that indicates a location after all previous materials 
	==   and meets the classes alignment requirements.
	== This offset will be added to the pointer that indicates the start of the buffer which will only be known at runtime.
	==
	== The dAbstractArray class should be told to only call the destructor of the materials, not delete which will corrupt memory
	==== ==== ==== ==== ==== ==== ==== ==== ==== ====
	*/



	MaterialCollectionFactory() = default;

	size_t QueueLambert(glm::vec3 albedo            ) { MaterialParams p { .lambert = { LAMBERTIAN, albedo       } }; queued_materials.push_back(p); return queued_materials.size() - 1; }
	size_t   QueueMetal(glm::vec3 albedo, float fuzz) { MaterialParams p { .metal   = {      METAL, albedo, fuzz } }; queued_materials.push_back(p); return queued_materials.size() - 1; }
	size_t  QueueDielec(glm::vec3 albedo, float  ior) { MaterialParams p { .dielec  = { DIELECTRIC, albedo,  ior } }; queued_materials.push_back(p); return queued_materials.size() - 1; }

	MaterialCollection MakeCollection() {
		darray<MaterialParams> params(queued_materials);
		auto material_factory = dobj<MaterialFactory>::Make(params.getPtr());

		dAbstractArray<Material> materials(queued_materials.size());
		materials.MakeOnDeviceFactory(queued_materials.size(), 0, 0, material_factory.getPtr());
	}
};
#endif

class SceneBook1FinaleFactory {
public:
	//enum MaterialID { LAMBERTIAN, METAL, DIELECTRIC };
	//struct MaterialParams {
	//	MaterialID matID;
	//	union {
	//		struct { glm::vec3 albedo; }				lambert;
	//		struct { glm::vec3 albedo; float fuzz; }	  metal;
	//		struct { glm::vec3 albedo; float  ior; }	 dielec;
	//	};

	//	static MaterialParams MakeLambert(glm::vec3 albedo            ) { MaterialParams p; p.matID = LAMBERTIAN; p.lambert = { albedo       }; return p; }
	//	static MaterialParams   MakeMetal(glm::vec3 albedo, float fuzz) { MaterialParams p; p.matID =      METAL; p.metal   = { albedo, fuzz }; return p; }
	//	static MaterialParams  MakeDielec(glm::vec3 albedo, float  ior) { MaterialParams p; p.matID = DIELECTRIC; p.dielec  = { albedo,  ior }; return p; }


	//	__device__ Material* MakeMaterial() const {
	//		switch (matID) {
	//		case LAMBERTIAN: return new LambertianAbstract(lambert.albedo);
	//		case      METAL: return new      MetalAbstract(metal.albedo, metal.fuzz);
	//		case DIELECTRIC: return new DielectricAbstract(dielec.albedo, dielec.ior);
	//		}
	//	}
	//};

	//class d_MaterialFactory {
	//	MaterialParams* p{};
	//public:
	//	__host__ __device__ d_MaterialFactory(MaterialParams* p) : p(p) {}
	//	__device__ Material* operator()(size_t index) const { return p[index].MakeMaterial(); }
	//};

	struct SphereParams {
		glm::vec3 origin;
		float radius;
		Material* mat_ptr;
		__device__ Hittable* MakeSphere() const { return new SphereHittable(origin, radius, mat_ptr); }
	};

	class d_SphereFactory {
		SphereParams* p{};
		//Material** mats;
	public:
		__host__ __device__ d_SphereFactory(SphereParams* p) : p(p) {}
		__device__ Hittable* operator()(size_t index) const { return p[index].MakeSphere(); }
	};

private:
	void _make_sphere(int a, int b) {
	#define rnd (host_rnd.next())

		float choose_mat = rnd;
		glm::vec3 pos(a + 0.9f * rnd, 0.2f, b + 0.9f * rnd);

		if (glm::length(pos - glm::vec3(4, 0.2f, 0)) > 0.9f) {
			if (choose_mat < 0.8f) {
				// diffuse
				glm::vec3 albedo = glm::vec3(rnd, rnd, rnd) * glm::vec3(rnd, rnd, rnd);
				
				auto material = dobj<LambertianAbstract>::Make(albedo);
				sphere_params.push_back({ pos, 0.2f, material.getPtr()});
				materials.push_back(std::move(material));
				//material_params.push_back(MaterialParams::MakeLambert(albedo));
			}
			else if (choose_mat < 0.95f) {
				// metal
				glm::vec3 albedo = glm::vec3(rnd, rnd, rnd) * 0.5f + 0.5f;
				float fuzz = rnd * 0.5f;

				auto material = dobj<MetalAbstract>::Make(albedo, fuzz);
				sphere_params.push_back({ pos, 0.2f, material.getPtr()});
				materials.push_back(std::move(material));
				//material_params.push_back(MaterialParams::MakeMetal(albedo, fuzz));
			}
			else {
				// glass
				auto material = dobj<DielectricAbstract>::Make(glm::vec3(1.0f), 1.5f);
				sphere_params.push_back({ pos, 0.2f, material.getPtr()});
				materials.push_back(std::move(material));
				//material_params.push_back(MaterialParams::MakeDielec(glm::vec3(1.0f), 1.5f));
			}
		}
	}

	void _populate_world() {
		// ground sphere
		auto ground_mat = dobj<LambertianAbstract>::Make(glm::vec3(0.5f));
		sphere_params.push_back({ glm::vec3(0,-1000,0), 1000, ground_mat.getPtr() });
		materials.push_back(std::move(ground_mat));
		//material_params.push_back(MaterialParams::MakeLambert(glm::vec3(0.5f)));

		for (int a = -11; a < 11; a++) {
			for (int b = -11; b < 11; b++) {
				_make_sphere(a, b);
			}
		}

		auto center_mat = dobj<DielectricAbstract>::Make(glm::vec3(1.0f), 1.5f);
		sphere_params.push_back({ glm::vec3(0,1,0),1, center_mat.getPtr()});
		materials.push_back(std::move(center_mat));
		//material_params.push_back(MaterialParams::MakeDielec(glm::vec3(1.0f), 1.5f));

		auto left_mat = dobj<LambertianAbstract>::Make(glm::vec3(0.4f, 0.2f, 0.1f));
		sphere_params.push_back({ glm::vec3(-4,1,0),1,left_mat.getPtr()});
		materials.push_back(std::move(left_mat));
		//material_params.push_back(MaterialParams::MakeLambert(glm::vec3(0.4f, 0.2f, 0.1f)));

		auto right_mat = dobj<MetalAbstract>::Make(glm::vec3(0.7f, 0.6f, 0.5f), 0);
		sphere_params.push_back({ glm::vec3(4,1,0),1,right_mat.getPtr()});
		materials.push_back(std::move(right_mat));
		//material_params.push_back(MaterialParams::MakeMetal(glm::vec3(0.7f, 0.6f, 0.5f), 0));
	}

	

	//std::vector<MaterialParams> material_params{};
	std::vector<dobj<Material>> materials;
	std::vector<  SphereParams> sphere_params;
	cuHostRND host_rnd{ 512, 1984 };

public:

	static _SceneDescription MakeScene() {
		SceneBook1FinaleFactory factory{};

		factory._populate_world();

		//darray<MaterialParams> d_mat_params(factory.material_params);
		//d_MaterialFactory material_factory(d_mat_params.getPtr());
		//auto d_mat_factory = dobj<d_MaterialFactory>::Make(material_factory);

		//dAbstractArray<Material> materials(factory.material_params.size());
		//materials.MakeOnDeviceFactory<d_MaterialFactory>(factory.material_params.size(), 0, 0, d_mat_factory.getPtr());


		darray<SphereParams> d_sphere_params(factory.sphere_params);
		d_SphereFactory sphere_factory(d_sphere_params.getPtr());
		auto d_sphere_factory = dobj<d_SphereFactory>::Make(sphere_factory);

		dAbstractArray<Hittable, true> sphere_list(factory.sphere_params.size());
		sphere_list.MakeOnDeviceFactory<d_SphereFactory>(factory.sphere_params.size(), 0, 0, d_sphere_factory.getPtr());

		
		auto world_list = dobj<HittableList>::Make(sphere_list.getDeviceArrayPtr(), sphere_list.getLength());

		return _SceneDescription {
			std::move(factory.materials),
			std::move(sphere_list),
			std::move(world_list)
		};
	}
};

#endif

FirstApp FirstApp::MakeApp() {
	uint32_t _width = 1280;
	uint32_t _height = 720;

	glm::vec3 lookfrom(13, 2, 3);
	glm::vec3 lookat(0, 0, 0);
	glm::vec3 up(0, 1, 0);
	float fov = 20.0f;
	float aspect = _width / (float)_height;
	PinholeCamera cam = PinholeCamera(lookfrom, lookat, up, fov, aspect);

	_SceneDescription scene_desc = SceneBook1FinaleFactory::MakeScene();

	Renderer renderer = Renderer::MakeRenderer(_width, _height, 8, 8, cam, scene_desc.world_list.getPtr());

	glm::vec4* host_output_framebuffer{};
	CUDA_ASSERT(hipHostMalloc(&host_output_framebuffer, sizeof(glm::vec4) * _width * _height));

	return FirstApp(M{
		_width,
		_height,
		cam,
		host_output_framebuffer,
		std::move(renderer),
		std::move(scene_desc),
	});
}
FirstApp::~FirstApp() {
	CUDA_ASSERT(hipHostFree(m.host_output_framebuffer));
}

void write_renderbuffer_png(std::string filepath, uint32_t width, uint32_t height, glm::vec4* data);
void FirstApp::Run() {
	m.renderer.Render();
	m.renderer.DownloadRenderbuffer(m.host_output_framebuffer);
	write_renderbuffer_png("../renders/test_040.png"s, m.render_width, m.render_height, m.host_output_framebuffer);
}

void write_renderbuffer_png(std::string filepath, uint32_t width, uint32_t height, glm::vec4* data) {
	//uint8_t* output_image_data = new uint8_t[width * height * 4];
	std::vector<uint8_t> output_image_data(width * height * 4);
	for (uint32_t i = 0; i < width * height; i++) {
		output_image_data.push_back(static_cast<uint8_t>(data[i][0] * 255.999f));
		output_image_data.push_back(static_cast<uint8_t>(data[i][1] * 255.999f));
		output_image_data.push_back(static_cast<uint8_t>(data[i][2] * 255.999f));
		output_image_data.push_back(static_cast<uint8_t>(data[i][3] * 255.999f));
	}

	stbi_flip_vertically_on_write(true);
	stbi_write_png(filepath.c_str(), width, height, 4, output_image_data.data(), sizeof(uint8_t) * width * 4);
	//delete[] output_image_data;
}