#include "FirstApp.cuh"

#include <inttypes.h>
#include <string>
using namespace std::string_literals;
#include <iostream>
#include <vector>
#include <algorithm>
#include <tuple>

#include <glm/glm.hpp>
#include <hip/hip_runtime.h>
#include <stb/stb_image_write.h>

#include "cuError.h"

#include "dobj.cuh"
#include "darray.cuh"
#include "dAbstracts.cuh"

#include "hittable.cuh"
#include "SphereHittable.cuh"
#include "HittableList.cuh"
#include "bvh_node.cuh"

#include "material.cuh"
#include "cu_Materials.cuh"

#include "texture.cuh"
#include "cu_Textures.cuh"

#include "cu_Cameras.cuh"
#include "Renderer.cuh"

#include "cuHostRND.h"


void SceneBook1::_delete() {
	CUDA_ASSERT(hipFree(world));
	CUDA_ASSERT(hipFree(hittable_list));
}

void SceneBook1::Factory::_populate_world() {
	
	Sphere ground_sphere = Sphere(glm::vec3(0, -1000, 0), 1000.0f);
	LambertianAbstract<Sphere>* ground_mat = newOnDevice<LambertianAbstract<Sphere>>(glm::vec3(0.5f));
	SphereHandle ground_handle = SphereHandle::MakeSphere(ground_sphere, ground_mat);
	world_bounds += ground_handle.getBounds();
	sphere_handles.push_back(std::move(ground_handle));

	for (int a = -11; a < 11; a++) {
		for (int b = -11; b < 11; b++) {

			#define rnd host_rnd.next()

			float choose_mat = rnd;
			glm::vec3 center(a + rnd, 0.2f, b + rnd);

			if (choose_mat < 0.8f) {
				auto material = newOnDevice<LambertianAbstract<MovingSphere>>(glm::vec3(rnd * rnd, rnd * rnd, rnd * rnd));
				glm::vec3 center1 = center + glm::vec3(0, rnd * 0.5f, 0);
				auto moving_sphere = MovingSphere(center, center1, 0.2f);
				auto handle = SphereHandle::MakeMovingSphere(moving_sphere, material);
				world_bounds += handle.getBounds();
				sphere_handles.push_back(std::move(handle));
			}
			else if (choose_mat < 0.95f) {
				auto material = newOnDevice<MetalAbstract<Sphere>>(glm::vec3(0.5f * (1.0f + rnd), 0.5f * (1.0f + rnd), 0.5f * (1.0f + rnd)), 0.5f * rnd);
				auto sphere = Sphere(center, 0.2f);
				auto handle = SphereHandle::MakeSphere(sphere, material);
				world_bounds += handle.getBounds();
				sphere_handles.push_back(std::move(handle));
			}
			else {
				auto material = newOnDevice<DielectricAbstract<Sphere>>(glm::vec3(1.0f), 1.5f);
				auto sphere = Sphere(center, 0.2f);
				auto handle = SphereHandle::MakeSphere(sphere, material);
				world_bounds += handle.getBounds();
				sphere_handles.push_back(std::move(handle));
			}
		}
	}

	Sphere center_sphere = Sphere(glm::vec3(0, 1, 0), 1);
	DielectricAbstract<Sphere>* center_mat = newOnDevice<DielectricAbstract<Sphere>>(glm::vec3(1.0f), 1.5f);
	auto center_handle = SphereHandle::MakeSphere(center_sphere, center_mat);
	world_bounds += center_handle.getBounds();
	sphere_handles.push_back(std::move(center_handle));

	Sphere left_sphere = Sphere(glm::vec3(-4, 1, 0), 1);
	LambertianAbstract<Sphere>* left_mat = newOnDevice<LambertianAbstract<Sphere>>(glm::vec3(0.4f, 0.2f, 0.1f));
	auto left_handle = SphereHandle::MakeSphere(left_sphere, left_mat);
	world_bounds += left_handle.getBounds();
	sphere_handles.push_back(std::move(left_handle));

	Sphere right_sphere = Sphere(glm::vec3(4, 1, 0), 1);
	MetalAbstract<Sphere>* right_mat = newOnDevice<MetalAbstract<Sphere>>(glm::vec3(0.7f, 0.6f, 0.5f), 0);
	auto right_handle = SphereHandle::MakeSphere(right_sphere, right_mat);
	world_bounds += right_handle.getBounds();
	sphere_handles.push_back(std::move(right_handle));
}

SceneBook1 SceneBook1::Factory::MakeScene() {

	printf("Populating world... ");
	_populate_world();
	printf("done.\n");

	printf("Building world's HittableList... ");
	std::vector<const Hittable*> hittable_vec{};
	for (int i = 0; i < sphere_handles.size(); i++) {
		const Hittable* ptr = sphere_handles[i].getHittablePtr();
		hittable_vec.push_back(ptr);
	}

	CUDA_ASSERT(hipMalloc((void**)&hittable_list, sizeof(Hittable*) * hittable_vec.size()));
	CUDA_ASSERT(hipMemcpy(hittable_list, hittable_vec.data(), sizeof(Hittable*) * hittable_vec.size(), hipMemcpyHostToDevice));
	world = newOnDevice<HittableList>(
		const_cast<const Hittable**>(hittable_list),
		(int)hittable_vec.size(),
		world_bounds
	);
	printf("done.\n");

	SceneBook1 scene;

	scene.world_bounds = world_bounds;
	scene.world = world;
	scene.hittable_list = hittable_list;
	scene.sphere_handles = std::move(sphere_handles);

	return scene;
}

SceneBook1::SceneBook1(SceneBook1&& scene) {
	world_bounds = scene.world_bounds;
	
	world = scene.world;
	scene.world = nullptr;

	hittable_list = scene.hittable_list;
	scene.hittable_list = nullptr;

	sphere_handles = std::move(scene.sphere_handles);
}

SceneBook1& SceneBook1::operator=(SceneBook1&& scene) {
	_delete();

	world_bounds = scene.world_bounds;

	world = scene.world;
	scene.world = nullptr;

	hittable_list = scene.hittable_list;
	scene.hittable_list = nullptr;

	sphere_handles = std::move(scene.sphere_handles);

	return *this;
}

SceneBook1::~SceneBook1() {
	_delete();
}


FirstApp FirstApp::MakeApp() {
	uint32_t _width = 1280;
	uint32_t _height = 720;

	printf("Building MotionBlurCamera object... ");
	glm::vec3 lookfrom(13, 2, 3);
	glm::vec3 lookat(0, 0, 0);
	glm::vec3 up(0, 1, 0);
	float fov = 30.0f;
	float aspect = _width / (float)_height;
	MotionBlurCamera cam(lookfrom, lookat, up, fov, aspect, 0.1f, 1.0f);
	printf("done.\n");

	printf("Building SceneBook1 object...\n");
	SceneBook1::Factory scene_factory{};
	SceneBook1 scene_desc = scene_factory.MakeScene();
	printf("SceneBook1 object built.\n");
		
	printf("Making Renderer object...\n");
	Renderer renderer = Renderer::MakeRenderer(_width, _height, 8, 12, cam, scene_desc.getWorldPtr());
	printf("Renderer object built.\n");

	glm::vec4* host_output_framebuffer{};
	printf("Allocating host framebuffer... ");
	CUDA_ASSERT(hipHostMalloc(&host_output_framebuffer, sizeof(glm::vec4) * _width * _height));
	printf("done.\n");

	return FirstApp(M{
		_width,
		_height,
		cam,
		host_output_framebuffer,
		std::move(renderer),
		std::move(scene_desc),
	});
}
FirstApp::~FirstApp() {
	printf("Freeing host framebuffer allocation... ");
	CUDA_ASSERT(hipHostFree(m.host_output_framebuffer));
	printf("done.\n");
}

void write_renderbuffer(std::string filepath, uint32_t width, uint32_t height, glm::vec4* data);
void FirstApp::Run() {
	printf("Rendering scene...\n");
	m.renderer.Render();
	printf("Scene rendered.\n");

	printf("Downloading render to host framebuffer... ");
	m.renderer.DownloadRenderbuffer(m.host_output_framebuffer);
	printf("done.\n");

	printf("Writing render to disk... ");
	write_renderbuffer("../renders/Book 2/test_011.jpg"s, m.render_width, m.render_height, m.host_output_framebuffer);
	printf("done.\n");
}

void write_renderbuffer(std::string filepath, uint32_t width, uint32_t height, glm::vec4* data) {
	//uint8_t* output_image_data = new uint8_t[width * height * 4];
	std::vector<uint8_t> output_image_data;
	output_image_data.reserve(width * height * 3);
	for (uint32_t i = 0; i < width * height; i++) {
		output_image_data.push_back(static_cast<uint8_t>(data[i][0] * 255.999f));
		output_image_data.push_back(static_cast<uint8_t>(data[i][1] * 255.999f));
		output_image_data.push_back(static_cast<uint8_t>(data[i][2] * 255.999f));
		//output_image_data.push_back(static_cast<uint8_t>(data[i][3] * 255.999f));
	}

	stbi_flip_vertically_on_write(true);
	stbi_write_jpg(filepath.c_str(), width, height, 3, output_image_data.data(), 95);
	//delete[] output_image_data;
}