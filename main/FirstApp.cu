#include "FirstApp.cuh"

#include <hiprand.h>

#include <stb/stb_image_write.h>

class Scene1Factory {
public:

	class d_MatFactory {
	public:
		__device__ Material* operator()(size_t input) const {
			switch (input) {
			case 0: return new LambertianAbstract(glm::vec3(0.8f, 0.8f, 0.0f));
			case 1: return new LambertianAbstract(glm::vec3(0.1f, 0.2f, 0.5f));
			case 2: return new DielectricAbstract(glm::vec3(1.0f, 1.0f, 1.0f), 1.5f);
			case 3: return new      MetalAbstract(glm::vec3(0.8f, 0.6f, 0.2f), 0.0f);
			default: return nullptr;
			}
		}
	};

	class d_SphereFactory {
		Material** mat_ptrs{};
	public:
		__device__ d_SphereFactory(Material** ptr2) : mat_ptrs(ptr2) {}
		__device__ Hittable* operator()(size_t input) const {
			switch (input) {
			case 0: return new Sphere(glm::vec3( 0.0f, -100.5f, -1.0f), 100.0f, mat_ptrs[0]);
			case 1: return new Sphere(glm::vec3( 0.0f,    0.0f, -1.0f),   0.5f, mat_ptrs[1]);
			case 2: return new Sphere(glm::vec3(-1.0f,    0.0f, -1.0f),   0.5f, mat_ptrs[2]);
			case 3: return new Sphere(glm::vec3(-1.0f,    0.0f, -1.0f),  -0.4f, mat_ptrs[2]);
			case 4: return new Sphere(glm::vec3( 1.0f,    0.0f, -1.0f),   0.5f, mat_ptrs[3]);
			default: return nullptr;
			}
		}
	};

	Scene1Factory(
		HandledDeviceAbstractArray<Material>** materials,
		HandledDeviceAbstractArray<Hittable>** spheres,
		HandledDeviceAbstract<HittableList>** world_list
	) {
		HandledDeviceAbstract<d_MatFactory> matFact{};
		matFact.MakeOnDevice<d_MatFactory>();
		
		(*materials) = new HandledDeviceAbstractArray<Material>(4);
		(*materials)->MakeOnDeviceFactoryPtr<d_MatFactory>(4, 0, 0, matFact.getPtr());

		HandledDeviceAbstract<d_SphereFactory> sphereFact((*materials)->getDeviceArrayPtr());
		(*spheres) = new HandledDeviceAbstractArray<Hittable>(5);
		(*spheres)->MakeOnDeviceFactoryPtr<d_SphereFactory>(5, 0, 0, sphereFact.getPtr());

		(*world_list) = new HandledDeviceAbstract<HittableList>((*spheres)->getDeviceArrayPtr(), 5);
	}
};

class SceneBook1FinaleFactory {

	static std::vector<float> _makeNUniforms(size_t N, size_t seed) {
		hiprandGenerator_t gen;
		float* d_rnd_uniforms{};
		hipMalloc(&d_rnd_uniforms, sizeof(float) * N);

		hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_XORWOW);
		hiprandSetPseudoRandomGeneratorSeed(gen, seed);
		hiprandGenerateUniform(gen, d_rnd_uniforms, N);

		std::vector<float> rnds{};
		rnds.resize(N);
		hipMemcpy(rnds.data(), d_rnd_uniforms, sizeof(float) * N, hipMemcpyDeviceToHost);

		hiprandDestroyGenerator(gen);
		hipFree(d_rnd_uniforms);
		return rnds;
	}

	template <typename T>
	static T* _copyToDevice(const std::vector<T>& v) {
		T* d_ptr{};
		CUDA_ASSERT(hipMalloc(&d_ptr, sizeof(T) * v.size()));
		CUDA_ASSERT(hipMemcpy(d_ptr, v.data(), sizeof(T) * v.size(), hipMemcpyHostToDevice));
		return d_ptr;
	}

	enum MatIdx { lambert, metal, dielec };
	struct LambertParams { glm::vec3 albedo; };
	struct MetalParams   { glm::vec3 albedo; float   fuzz; };
	struct DielecParams  { glm::vec3 albedo; float    ior; };
	struct SphereParams  { glm::vec3 origin; float radius; MatIdx mat_type; size_t mat_index; };

	void _makeSphere(int a, int b, std::vector<float>& rnds, int& rnd_offset) {
	#define rnd (rnds[rnd_offset++])

		float choose_mat = rnd;
		glm::vec3 pos(a + 0.9f * rnd, 0.2f, b + 0.9f * rnd);

		if (glm::length(pos - glm::vec3(4, 0.2f, 0)) > 0.9f) {
			if (choose_mat < 0.8f) {
				// diffse
				glm::vec3 albedo = glm::vec3(rnd, rnd, rnd) * glm::vec3(rnd, rnd, rnd);
				sphere_params.push_back({ pos, 0.2f, lambert, lambert_params.size() });
				lambert_params.push_back({ albedo });
			}
			else if (choose_mat < 0.95f) {
				// metal
				glm::vec3 albedo = glm::vec3(rnd, rnd, rnd) * 0.5f + 0.5f;
				float fuzz = rnd * 0.5f;
				sphere_params.push_back({ pos, 0.2f, metal, metal_params.size() });
				metal_params.push_back({ albedo, fuzz });
			}
			else {
				// glass
				sphere_params.push_back({ pos, 0.2f, dielec, dielec_params.size() });
				dielec_params.push_back({ glm::vec3(1.0f), 1.5f });
			}
		}
	}

	void _populateWorld() {
		int rnd_offset = 0;
		auto rnds = _makeNUniforms(4096, 1984);

		// ground sphere
		sphere_params.push_back({ glm::vec3(0,-1000,0), 1000, lambert, lambert_params.size()});
		lambert_params.push_back({ glm::vec3(0.5f) });

		for (int a = -11; a < 11; a++) {
			for (int b = -11; b < 11; b++) {
				_makeSphere(a, b, rnds, rnd_offset);
			}
		}

		sphere_params.push_back({ glm::vec3(0, 1, 0), 1.0f, dielec, dielec_params.size() });
		dielec_params.push_back({ glm::vec3(1.0f), 1.5f });

		sphere_params.push_back({ glm::vec3(-4, 1, 0), 1.0f, lambert, lambert_params.size() });
		lambert_params.push_back({ glm::vec3(0.4f, 0.2f, 0.1f) });

		sphere_params.push_back({ glm::vec3(4, 1, 0), 1.0f, metal, metal_params.size() });
		metal_params.push_back({ glm::vec3(0.7f, 0.6f, 0.5f), 0.0f });
	}

	std::vector<LambertParams> lambert_params{};
	std::vector<MetalParams> metal_params{};
	std::vector<DielecParams> dielec_params{};
	std::vector<SphereParams> sphere_params{};

public:

	class d_LambertFactory {
		LambertParams* p{};
	public:
		__host__ __device__ d_LambertFactory(LambertParams* p) : p(p) {}
		__device__ Material* operator()(size_t index) const {
			LambertParams& p2 = p[index];
			return new LambertianAbstract(p2.albedo);
		}
	};
	class d_MetalFactory {
		MetalParams* p{};
	public:
		__host__ __device__ d_MetalFactory(MetalParams* p) : p(p) {}
		__device__ Material* operator()(size_t index) const {
			MetalParams& p2 = p[index];
			return new MetalAbstract(p2.albedo, p2.fuzz);
		}
	};
	class d_DielecFactory {
		DielecParams* p{};
	public:
		__host__ __device__ d_DielecFactory(DielecParams* p) : p(p) {}
		__device__ Material* operator()(size_t index) const {
			DielecParams& p2 = p[index];
			return new DielectricAbstract(p2.albedo, p2.ior);
		}
	};
	class d_SphereFactory {
		SphereParams* p{};
		Material** mats;
		int lambert_offset{}, metal_offset{}, dielec_offset{};

		__device__ int _getMatOffset(MatIdx idx) const {
			switch (idx) {
			case lambert: return lambert_offset;
			case   metal: return   metal_offset;
			case  dielec: return  dielec_offset;
			}
		}

	public:
		__host__ __device__ d_SphereFactory(SphereParams* p, Material** mats, int lambert_offset, int metal_offset, int dielec_offset)
			: p(p), mats(mats), lambert_offset(lambert_offset), metal_offset(metal_offset), dielec_offset(dielec_offset) {}
		__device__ Hittable* operator()(size_t index) const {
			SphereParams& p2 = p[index];
			int mat_offset = p2.mat_index + _getMatOffset(p2.mat_type);
			return new Sphere(p2.origin, p2.radius, mats[mat_offset]);
		}
	};

	SceneBook1FinaleFactory(
		std::unique_ptr<HandledDeviceAbstractArray<Material>>& materials,
		std::unique_ptr<HandledDeviceAbstractArray<Hittable>>& sphere_list,
		std::unique_ptr<HandledDeviceAbstract<HittableList>>& world_list
	) {
		_populateWorld();

		LambertParams* d_lambert_params = _copyToDevice(lambert_params);
		d_LambertFactory lambert_factory(d_lambert_params);

		MetalParams* d_metal_params = _copyToDevice(metal_params);
		d_MetalFactory metal_factory(d_metal_params);

		DielecParams* d_dielec_params = _copyToDevice(dielec_params);
		d_DielecFactory dielec_factory(d_dielec_params);

		size_t lambert_offset = 0;
		size_t   metal_offset = lambert_offset + lambert_params.size();
		size_t  dielec_offset =   metal_offset +   metal_params.size();

		materials = std::make_unique<HandledDeviceAbstractArray<Material>>(sphere_params.size());
		materials->MakeOnDeviceFactory<d_LambertFactory>(lambert_params.size(), lambert_offset, 0, lambert_factory);
		materials->MakeOnDeviceFactory<d_MetalFactory  >(  metal_params.size(),   metal_offset, 0,   metal_factory);
		materials->MakeOnDeviceFactory<d_DielecFactory >( dielec_params.size(),  dielec_offset, 0,  dielec_factory);

		CUDA_ASSERT(hipFree(d_lambert_params));
		CUDA_ASSERT(hipFree(d_metal_params  ));
		CUDA_ASSERT(hipFree(d_dielec_params ));


		SphereParams* d_sphere_params = _copyToDevice(sphere_params);
		d_SphereFactory sphere_factory(d_sphere_params, materials->getDeviceArrayPtr(), lambert_offset, metal_offset, dielec_offset);

		sphere_list = std::make_unique<HandledDeviceAbstractArray<Hittable>>(sphere_params.size());
		sphere_list->MakeOnDeviceFactory<d_SphereFactory>(sphere_params.size(), 0, 0, sphere_factory);

		CUDA_ASSERT(hipFree(d_sphere_params));


		world_list = std::make_unique<HandledDeviceAbstract<HittableList>>(sphere_list->getDeviceArrayPtr(), sphere_list->getSize());
	}
};

FirstApp::FirstApp() {
	render_width = 1280;
	render_height = 720;

	glm::vec3 lookfrom(13, 2, 3);
	glm::vec3 lookat(0, 0, 0);
	glm::vec3 up(0, 1, 0);
	float fov = 20.0f;
	float aspect = render_width / (float)render_height;
	cam = PinholeCamera(lookfrom, lookat, up, fov, aspect);

	SceneBook1FinaleFactory(sphere_materials, world_sphere_list, world_list);

	renderer = std::make_unique<Renderer>(render_width, render_height, 1024, 32, cam, world_list->getPtr());

	CUDA_ASSERT(hipHostMalloc(&host_output_framebuffer, sizeof(glm::vec4) * render_width * render_height));
}
FirstApp::~FirstApp() {
	CUDA_ASSERT(hipHostFree(host_output_framebuffer));
}

void write_renderbuffer_png(std::string filepath, uint32_t width, uint32_t height, glm::vec4* data);
void FirstApp::Run() {
	renderer->Render();
	renderer->DownloadRenderbuffer(host_output_framebuffer);
	write_renderbuffer_png("../renders/test_039.png"s, render_width, render_height, host_output_framebuffer);
}

void write_renderbuffer_png(std::string filepath, uint32_t width, uint32_t height, glm::vec4* data) {
	uint8_t* output_image_data = new uint8_t[width * height * 4];
	for (uint32_t i = 0; i < width * height; i++) {
		output_image_data[i * 4 + 0] = static_cast<uint8_t>(data[i][0] * 255.999f);
		output_image_data[i * 4 + 1] = static_cast<uint8_t>(data[i][1] * 255.999f);
		output_image_data[i * 4 + 2] = static_cast<uint8_t>(data[i][2] * 255.999f);
		output_image_data[i * 4 + 3] = static_cast<uint8_t>(data[i][3] * 255.999f);
	}

	stbi_flip_vertically_on_write(true);
	stbi_write_png(filepath.c_str(), width, height, 4, output_image_data, sizeof(uint8_t) * width * 4);
	delete[] output_image_data;
}